#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "include/kdac_cuda.h"
#include "include/matrix.h"
#include "include/vector.h"
#include "include/gpu_util.h"
#include "include/util.h"
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <unistd.h>
#include <stdexcept>
#include <ctime>

namespace Nice {

template <typename T>
T* CUDAMallocAndCpy(const Matrix<T> &mat) {
  GpuUtil<T> *util = GpuUtil<T>::GetInstance();
  int n = mat.cols() * mat.rows();
  const T *h_mat = &mat(0);
  T *d_mat;
  util -> SetupMem(&d_mat, h_mat, n);
  std::cout << "allocating " << n * sizeof(T) << " bytes." << std::endl;
  return d_mat;
}
// Template explicit instantiation
template
float* CUDAMallocAndCpy<float>(const Matrix<float> &mat);
template
double* CUDAMallocAndCpy<double>(const Matrix<double> &mat);


template <typename T>
T* CUDAMallocAndCpy(const Vector <T> &vec) {
  GpuUtil<T> *util = GpuUtil<T>::GetInstance();
  int n = vec.size();
  const T *h_vec = &vec(0);
  T *d_vec;
  util -> SetupMem(&d_vec, h_vec, n);
  std::cout << "allocating " << n * sizeof(T) << " bytes." << std::endl;
  return d_vec;
}

template
float* CUDAMallocAndCpy<float>(const Vector<float> &vec);
template
double* CUDAMallocAndCpy<double>(const Vector<double> &vec);

// Position for Column-Major index
#define IDXC(i,j,ld) (((j)*(ld))+(i))
// Position for Row-Major index
#define IDXR(i,j,ld) (((i)*(ld))+(j))

template <typename T>
__global__ void GPUGenAMatricesKernel
    (T *x_matrix_d, T *a_matrices_d, T *all_delta_ijs_d, int n, int d) {

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // This is to index an n x n matrix where each cell is a
  // d x d matrix. No matter what orientation (row or column) the
  // d x d matrix is, to find the starting location of the (i, j)
  // matrix, we just need to use the following to do so
  if (i < n && j < n) {
    T *a_ij_matrix = a_matrices_d + IDXR(i, j, n) * (d * d);
    T *delta_ij = all_delta_ijs_d + IDXR(i, j, n) * d;

    // x_matrix_d is column major
    for (int k = 0; k < d; k++) {
      delta_ij[k] = x_matrix_d[IDXC(i, k, n)] - x_matrix_d[IDXC(j, k, n)];
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0;
    const float beta = 0.0;

//  Each thread (i, j) generates a matrix Aij
    hipblasSgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                d, d, 1,
                &alpha, delta_ij, d,
                delta_ij, 1,
                &beta, a_ij_matrix, d);
  }

////  hipblasStatus_t hipblasSgemm(hipblasHandle_t handle,
////                             hipblasOperation_t transa,
////                             hipblasOperation_t transb,
////                             int m, int n, int k,
////                             const float *alpha, const float *A, int lda,
////                             const float *B, int ldb,
////                             const float *beta, float *C, int ldc)
}

// Explicit Instantiation
template
__global__ void GPUGenAMatricesKernel<float>(float *x_matrix_d,
                            float *a_matrices_d,
                            float *all_delta_ijs_d,
                            int n,
                            int d);



template<typename T>
void GPUGenAMatrices(T *x_matrix_d, T *a_matrices_d, int n, int d) {

  GpuUtil<T> *gpu_util = GpuUtil<T>::GetInstance();
  int size_delta_ijs = n * n * d;
//  T *all_delta_ijs = new T[size_delta_ijs];
  T *all_delta_ijs_d;
  gpu_util -> SetupMem(&all_delta_ijs_d, nullptr, size_delta_ijs, false);
  int block_size = 16;
  dim3 dim_block(block_size, block_size);
  dim3 dim_grid( (n-1) / block_size + 1, (n-1) / block_size + 1);
  GPUGenAMatricesKernel<<<dim_grid, dim_block>>>(x_matrix_d, a_matrices_d,
      all_delta_ijs_d, n, d);
//  gpu_util->SyncMem(all_delta_ijs_d, all_delta_ijs, size_delta_ijs);
//  for (int i = 0; i < n; i++) {
//    for (int j = 0; j < n; j++) {
//      T *delta_ij = all_delta_ijs + IDXR(i, j, n) * d;
//      std::cout << "delta (" << (i + 1) << ", " << (j + 1) << "): "
//                << std::endl;
//      util::PrintMatrix<T>(delta_ij, d, 1, false);
//    }
//  }
}

// Explicit Instantiation
template
void GPUGenAMatrices<float>(float *x_matrix_d,
                            float *a_matrices_d,
                            int n,
                            int d);

// Cannot instantiate it to double if I am using hipblasSgemm
// Only hipblasDgemm is for double
//template
//void GPUGenAMatrices<double>(double *x_matrix,
//                             double *a_matrices,
//                             double *delta_ijs,
//                             int n,
//                             int d);
//template <typename T>
//__global__ void GPUGenPhiCoeffKernel(T *x_matrix,
//                                     T *a_matrices,
//                                     T *delta_x_ijs,
//                                     T *waw_matrix,
//                                     T *waf_matrix,
//                                     T *faf_matrix,
//                                     T *w_l,
//                                     T *gradient,
//                                     int n,
//                                     int d) {
//  int i = blockIdx.y * blockDim.y + threadIdx.y;
//  int j = blockIdx.x * blockDim.x + threadIdx.x;
//  // This is to index an n x n matrix where each cell is a
//  // d x d matrix. No matter what orientation (row or column) the
//  // d x d matrix is, to find the starting location of the (i, j)
//  // matrix, we just need to use the following to do so
//  T *a_ij_matrix = a_matrices + (i * n + j) * d * d;
//  T *delta_x_ij = delta_x_ijs + (i * n + j) * d;
//  GenAMatrix(x_matrix, a_matrices, a_ij_matrix, delta_x_ij, i, j, n, d);
//}
//
//template<typename T>
//void GPUGenPhiCoeff(T *x_matrix, T *a_matrices, T *waw_matrix, T *waf_matrix,
//                    T *faf_matrix, T *w_l, T *gradient, int n, int d) {
//  std::cout << "in GPUGenPhiCoeff" << std::endl;
//
//  int block_size = 16;
//  dim3 dim_block(block_size, block_size);
//  dim3 dim_grid( (n-1) / block_size + 1, (n-1) / block_size + 1);
//  GPUGenPhiCoeffKernel<<<dim_grid, dim_block>>>(x_matrix, waw_matrix,
//      waf_matrix, faf_matrix, w_l, gradient, n d);
//}
//
//template
//void GPUGenPhiCoeff<float>(float *x_matrix,
//                           float *a_matrices,
//                           float *delta_x_ijs,
//                           float *waw_matrix,
//                           float *waf_matrix,
//                           float *faf_matrix,
//                           float *w_l,
//                           float *gradient,
//                           int n,
//                           int d);
//template
//void GPUGenPhiCoeff<double>(double *x_matrix,
//                            double *a_matrices,
//                            double *delta_x_ijs,
//                            double *waw_matrix,
//                            double *waf_matrix,
//                            double *faf_matrix,
//                            double *w_l,
//                            double *gradient,
//                            int n,
//                            int d);
}