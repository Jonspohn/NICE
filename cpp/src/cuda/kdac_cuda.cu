#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "include/kdac_cuda.h"
#include "include/gpu_util.h"

namespace Nice {

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

bool isPow2(unsigned int x) {
  return ((x & (x - 1)) == 0);
}

template <typename T>
__global__ void GPUGenDeltaKernel(const T *x_matrix_d,
                                      const int n,
                                      const int d,
                                      T *all_delta_ijs_d) {

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // This is to index an n x n matrix where each cell is a
  // d x d matrix. No matter what orientation (row or column) the
  // d x d matrix is, to find the starting location of the (i, j)
  // matrix, we just need to use the following to do so
  if (i < n && j < n) {
    T *delta_ij = all_delta_ijs_d + IDXR(i, j, n) * d;
    // x_matrix_d is column major
    for (int k = 0; k < d; k++)
      delta_ij[k] = x_matrix_d[IDXC(i, k, n)] - x_matrix_d[IDXC(j, k, n)];
  }
}

template <typename T>
__global__ void GPUGenAMatricesKernel(const T *x_matrix_d,
                                      const int n,
                                      const int d,
                                      T *a_matrices_d) {
  T *delta_ij = SharedMemory<T>();
  int tx = threadIdx.x;
  int i = blockIdx.y;
  int j = blockIdx.x;

  if (tx < d) {
    T *a_ij = a_matrices_d + IDXR(i, j, n) * (d * d);
    delta_ij[tx] = x_matrix_d[IDXC(i, tx, n)] - x_matrix_d[IDXC(j, tx, n)];
    __syncthreads();
    // thread tx calculates a whole row tx of the output matrix a_ij
    for (int col = 0; col < d; col++)
      a_ij[IDXC(tx, col, d)] = delta_ij[col] * delta_ij[tx];
  }
}

template <typename T>
__global__ void GPUGenPhiCoeffKernel(const T *w_l_d,
                                     const T *gradient_d,
                                     const T *a_matrices_d,
                                     const T *gamma_matrix_d,
                                     const int n,
                                     const int d,
                                     const T alpha,
                                     const T sqrt_one_minus_alpha,
                                     const T *gamma
                                     T *waw_matrix_d,
                                     T *waf_matrix_d,
                                     T *faf_matrix_d) {
  T *vec = SharedMemory<T>();
  T *waw = (T*)vec;
  T *waf = (T*)&waw[blockDim.x];
  T *faf = (T*)&waw[2*blockDim.x];

  int i = blockIdx.y;
  int j = blockIdx.x;
  int tx = threadIdx.x;
  const T *a_ij = a_matrices_d + IDXR(i, j, n) * (d * d);
  const T gamma_ij = gamma_matrix_d[IDXC(j, i, n)];

  waw[tx] = 0.0;
  waf[tx] = 0.0;
  faf[tx] = 0.0;


  if (tx < d) {
    // Each tx takes care of one row of matrix in order to have a
    // coalesced access pattern
    // Each time it aggreates a column
    for (int col = 0; col < d; col++) {
      waw[tx] += a_ij[IDXC(tx, col, d)] * w_l_d[col];
      waf[tx] += a_ij[IDXC(tx, col, d)] * gradient_d[col];
      faf[tx] += a_ij[IDXC(tx, col, d)] * gradient_d[col];
    }

    // This is the dot product
    waw[tx] = waw[tx] * w_l_d[tx];
    waf[tx] = waf[tx] * w_l_d[tx];
    faf[tx] = faf[tx] * gradient_d[tx];
  }
  __syncthreads();

  // Reduction for dot product
  for (unsigned int s = blockDim.x / 2; s > 0; s>>=1) {
    if (tx < s) {
      waw[tx] += waw[tx + s];
      waf[tx] += waf[tx + s];
      faf[tx] += faf[tx + s];
    }
    __syncthreads();
  }
//    if (tx < 8) {
//      vec[tx] += vec[tx + 8];
//      vec[tx] += vec[tx + 4];
//      vec[tx] += vec[tx + 2];
//      vec[tx] += vec[tx + 1];
//    }
//    __syncthreads();

    // Transposed access for better access pattern as waw matrix is column-major
  if (tx == 0) {
    waw_matrix_d[IDXC(j, i, n)] = waw[tx];
    waf_matrix_d[IDXC(j, i, n)] = waf[tx];
    faf_matrix_d[IDXC(j, i, n)] = faf[tx];
  }


  if (tx == 0) {


  }
}

template <typename T>
__global__ void GPUGenPhiKernel(const T alpha,
                                const T sqrt_one_minus_alpha,
                                const T denom,
                                const T *waw_matrix_d,
                                const T *waf_matrix_d,
                                const T *faf_matrix_d,
                                const T *gamma_matrix_d,
                                const int n,
                                const int d,
                                bool w_l_changed,
                                float *phi_of_alphas_d,
                                float *phi_of_zeros_d,
                                float *phi_of_zero_primes_d) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if ((i < n) && (j < n)) {
    T waw = waw_matrix_d[IDXC(i, j, n)];
    T waf = waf_matrix_d[IDXC(i, j, n)];
    T faf = faf_matrix_d[IDXC(i, j, n)];
    T gammaij = gamma_matrix_d[IDXC(i, j, n)];
    T kij = expf(denom * ((faf - waw) * (alpha*alpha) +
        2 * waf * sqrt_one_minus_alpha * alpha + waw));
    phi_of_alphas_d[IDXC(i, j, n)] = gammaij * kij;
    if(w_l_changed) {
      T kij = expf(denom * waw);
      phi_of_zeros_d[IDXC(i, j, n)] = gammaij * kij;
      phi_of_zero_primes_d[IDXC(i, j, n)] =
          gammaij * denom * 2 * waf * kij;
    }
  }
}

template <typename T>
__global__ void GPUGenPhiTransposeKernel(const T alpha,
                                const T sqrt_one_minus_alpha,
                                const T denom,
                                const T *waw_matrix_d,
                                const T *waf_matrix_d,
                                const T *faf_matrix_d,
                                const T *gamma_matrix_d,
                                const int n,
                                const int d,
                                bool w_l_changed,
                                float *phi_of_alphas_d,
                                float *phi_of_zeros_d,
                                float *phi_of_zero_primes_d) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if ((i < n) && (j < n)) {
    T waw = waw_matrix_d[IDXC(j, i, n)];
    T waf = waf_matrix_d[IDXC(j, i, n)];
    T faf = faf_matrix_d[IDXC(j, i, n)];
    T gammaij = gamma_matrix_d[IDXC(j, i, n)];
    T kij = expf(denom * ((faf - waw) * (alpha*alpha) +
        2 * waf * sqrt_one_minus_alpha * alpha + waw));
    phi_of_alphas_d[IDXC(j, i, n)] = gammaij * kij;
    if(w_l_changed) {
      T kij = expf(denom * waw);
      phi_of_zeros_d[IDXC(j, i, n)] = gammaij * kij;
      phi_of_zero_primes_d[IDXC(j, i, n)] =
          gammaij * denom * 2 * waf * kij;
    }
  }
}



// From CUDA SDK
template <typename T, unsigned int blockSize, bool nIsPow2>
__global__ void reduce_kernel(T *g_idata, T *g_odata, unsigned int n) {
  T *sdata = SharedMemory<T>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  T mySum = 0;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n)
  {
    mySum += g_idata[i];

    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + blockSize < n)
      mySum += g_idata[i+blockSize];

    i += gridSize;
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = mySum;
  __syncthreads();


  // do reduction in shared mem
  if ((blockSize >= 512) && (tid < 256))
  {
    sdata[tid] = mySum = mySum + sdata[tid + 256];
  }

  __syncthreads();

  if ((blockSize >= 256) &&(tid < 128))
  {
    sdata[tid] = mySum = mySum + sdata[tid + 128];
  }

  __syncthreads();

  if ((blockSize >= 128) && (tid <  64))
  {
    sdata[tid] = mySum = mySum + sdata[tid +  64];
  }

  __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
  if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
  // fully unroll reduction within a single warp
  if ((blockSize >=  64) && (tid < 32))
  {
    sdata[tid] = mySum = mySum + sdata[tid + 32];
  }

  __syncthreads();

  if ((blockSize >=  32) && (tid < 16))
  {
    sdata[tid] = mySum = mySum + sdata[tid + 16];
  }

  __syncthreads();

  if ((blockSize >=  16) && (tid <  8))
  {
    sdata[tid] = mySum = mySum + sdata[tid +  8];
  }

  __syncthreads();

  if ((blockSize >=   8) && (tid <  4))
  {
    sdata[tid] = mySum = mySum + sdata[tid +  4];
  }

  __syncthreads();

  if ((blockSize >=   4) && (tid <  2))
  {
    sdata[tid] = mySum = mySum + sdata[tid +  2];
  }

  __syncthreads();

  if ((blockSize >=   2) && ( tid <  1))
  {
    sdata[tid] = mySum = mySum + sdata[tid +  1];
  }

  __syncthreads();
#endif

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = mySum;
}


template<typename T>
void GPUGenAMatrices(const T *x_matrix_d,
                     const int n,
                     const int d,
                     T *a_matrices_d) {

  unsigned int block_size = nextPow2(d);
  int shared_mem_size = d * sizeof(T) * 2;

  dim3 dim_block(block_size, 1);
  dim3 dim_grid(n, n);
  GPUGenAMatricesKernel
      <<<dim_grid, dim_block, shared_mem_size>>>
      (x_matrix_d, n, d, a_matrices_d);
}

// Explicit Instantiation
template
void GPUGenAMatrices<float>(const float *x_matrix_d,
                            const int n,
                            const int d,
                            float *a_matrices_d);

template <typename T>
void GPUGenPhiCoeff(const T *w_l_d,
                    const T *gradient_d,
                    const T *a_matrices_d,
                    const int n,
                    const int d,
                    T *waw_matrix_d,
                    T *waf_matrix_d,
                    T *faf_matrix_d) {
  int block_size = (isPow2(d)) ? d : nextPow2(d);
  int shared_mem_size = 3 * block_size * sizeof(T);
  dim3 dim_block(block_size, 1);
  dim3 dim_grid(n, n);
  GPUGenPhiCoeffKernel
      <<<dim_grid, dim_block, shared_mem_size>>>
      (w_l_d, gradient_d, a_matrices_d, n, d,
          waw_matrix_d, waf_matrix_d, faf_matrix_d);
  CUDA_CALL(hipGetLastError());

}

template
void GPUGenPhiCoeff<float>(const float *w_l_d,
                           const float *gradient_d,
                           const float *a_matrices_d,
                           const int n,
                           const int d,
                           float *waw_matrix_d,
                           float *waf_matrix_d,
                           float *faf_matrix_d);


void GetNumBlocksAndThreads(int num_elements,
                            int max_blocks,
                            int max_threads,
                            int &blocks,
                            int &threads) {
  hipDeviceProp_t prop;
  int device;
  CUDA_CALL(hipGetDevice(&device));
  CUDA_CALL(hipGetDeviceProperties(&prop, device));

  threads = (num_elements < max_threads) ?
            nextPow2(num_elements) : max_threads;
  blocks = (num_elements + (threads * 2 - 1)) / (threads * 2);

  if ((float)threads*blocks > (float)prop.maxGridSize[0] *
      prop.maxThreadsPerBlock) {
    printf("Number of elements is too large\n");
    exit(1);
  }
  if (blocks > prop.maxGridSize[0]) {
    printf("Grid size <%d> excceeds the device capability <%d>, "
               "set block size as %d (original %d)\n",
           blocks, prop.maxGridSize[0], threads*2, threads);

    blocks /= 2;
    threads *= 2;
  }
  blocks = (max_blocks < blocks) ? max_blocks : blocks;
}

template <typename T>
void reduce(int num_elements, int num_threads, int num_blocks,
            T *input_data_d, T *output_data_d) {



  dim3 dim_block(num_threads, 1, 1);
  dim3 dim_grid(num_blocks, 1, 1);

  int shared_mem_size = (num_threads <= 32) ?
                        2 * num_threads * sizeof(T) : num_threads * sizeof(T);
  if (isPow2(num_elements)) {
    switch (num_threads) {
      case 512:
        reduce_kernel < T, 512, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 256:
        reduce_kernel < T, 256, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 128:
        reduce_kernel < T, 128, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 64:
        reduce_kernel < T, 64, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 32:
        reduce_kernel < T, 32, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 16:
        reduce_kernel < T, 16, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 8:
        reduce_kernel < T, 8, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 4:
        reduce_kernel < T, 4, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 2:
        reduce_kernel < T, 2, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 1:
        reduce_kernel < T, 1, true ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
    }
  } else {
    switch (num_threads) {
      case 512:
        reduce_kernel < T, 512, false
            ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 256:
        reduce_kernel < T, 256, false
            ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 128:
        reduce_kernel < T, 128, false
            ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 64:
        reduce_kernel < T, 64, false ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 32:
        reduce_kernel < T, 32, false ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 16:
        reduce_kernel < T, 16, false ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 8:
        reduce_kernel < T, 8, false ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 4:
        reduce_kernel < T, 4, false ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 2:
        reduce_kernel < T, 2, false ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
      case 1:
        reduce_kernel < T, 1, false ><<<dim_grid, dim_block, shared_mem_size>>>
        (input_data_d, output_data_d, num_elements);
        break;
    }
  }
}

template
void reduce<float>(int num_elements, int num_threads, int num_blocks,
            float *input_data_d, float *output_data_d);



template<typename T>
void GPUGenPhi(const T alpha,
               const T sqrt_one_minus_alpha,
               const T denom,
               const T *waw_matrix_d,
               const T *waf_matrix_d,
               const T *faf_matrix_d,
               const T *gamma_matrix_d,
               const int n,
               const int d,
               const bool w_l_changed,
               T *phi_of_alphas_in_d,
               T *phi_of_zeros_in_d,
               T *phi_of_zero_primes_in_d) {
  int block_size = 16;

  dim3 dim_block(block_size, block_size);
  // If matrix is n x m, then I need an m x n grid for contiguous
  // memory access
  dim3 dim_grid( (n-1) / block_size + 1, (n-1) / block_size + 1);

  GPUGenPhiTransposeKernel<<<dim_grid, dim_block>>>(alpha,
                                           sqrt_one_minus_alpha,
                                           denom,
                                           waw_matrix_d,
                                           waf_matrix_d,
                                           faf_matrix_d,
                                           gamma_matrix_d,
                                           n,
                                           d,
                                           w_l_changed,
                                           phi_of_alphas_in_d,
                                           phi_of_zeros_in_d,
                                           phi_of_zero_primes_in_d);

  // Check if error happens in kernel launch
  CUDA_CALL(hipGetLastError());

  int num_blocks = 0;
  int num_threads = 0;
  int num_elems = n * n;
  int max_blocks = 64;
  int max_threads = 256;
  GetNumBlocksAndThreads(num_elems,
                         max_blocks,
                         max_threads,
                         num_blocks,
                         num_threads);

  // Each block generates a partial sum
  T *phi_of_alphas_out_d = 0;
  CUDA_CALL(hipMalloc((void**) &phi_of_alphas_out_d,
                       num_blocks * sizeof(T)));
  reduce<T>(num_elems, num_threads, num_blocks,
            phi_of_alphas_in_d, phi_of_alphas_out_d);
  CUDA_CALL(hipGetLastError());
  T *phi_of_alphas_out_h = new T[n * n];
  CUDA_CALL(hipMemcpy(phi_of_alphas_out_h, phi_of_alphas_out_d,
                       num_blocks * sizeof(T), hipMemcpyDeviceToHost));
  T phi_of_alpha = 0;
  for (int i = 0; i < num_blocks; i++)
    phi_of_alpha += phi_of_alphas_out_h[i];
//  printf("phi(alpha) on gpu: %f\n", phi_of_alpha);
  CUDA_CALL(hipFree(phi_of_alphas_out_d));
  delete [] phi_of_alphas_out_h;

  if (w_l_changed) {
    T *phi_of_zeros_out_d = 0;
    T *phi_of_zero_primes_out_d = 0;
    CUDA_CALL(hipMalloc((void**) &phi_of_zeros_out_d,
                         num_blocks * sizeof(T)));
    CUDA_CALL(hipMalloc((void**) &phi_of_zero_primes_out_d,
                         num_blocks * sizeof(T)));
    reduce<T>(num_elems, num_threads, num_blocks,
              phi_of_zeros_in_d, phi_of_zeros_out_d);
    CUDA_CALL(hipGetLastError());
    reduce<T>(num_elems, num_threads, num_blocks,
              phi_of_zero_primes_in_d, phi_of_zero_primes_out_d);
    CUDA_CALL(hipGetLastError());
    T *phi_of_zeros_out_h = new T[n * n];
    T *phi_of_zero_primes_out_h = new T[n * n];
    CUDA_CALL(hipMemcpy(phi_of_zeros_out_h, phi_of_zeros_out_d,
                         num_blocks * sizeof(T), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(phi_of_zero_primes_out_h, phi_of_zero_primes_out_d,
                         num_blocks * sizeof(T), hipMemcpyDeviceToHost));
    T phi_of_zero = 0;
    T phi_of_zero_prime = 0;
    for (int i = 0; i < num_blocks; i++) {
      phi_of_zero += phi_of_zeros_out_h[i];
      phi_of_zero_prime += phi_of_zero_primes_out_h[i];
    }
//    printf("phi(0) on gpu: %f\n", phi_of_zero);
//    printf("phi(0)' on gpu: %f\n", phi_of_zero_prime);
    CUDA_CALL(hipFree(phi_of_zeros_out_d));
    CUDA_CALL(hipFree(phi_of_zero_primes_out_d));
    delete [] phi_of_zeros_out_h;
    delete [] phi_of_zero_primes_out_h;
  }
//  if (w_l_changed) {
//    reduce<T>(num_elems, num_threads, num_blocks,
//              phi_of_zeros_in_d, phi_of_zeros_out_d);
//    reduce<T>(num_elems, num_threads, num_blocks,
//              phi_of_zero_primes_in_d, phi_of_zero_primes_out_d);
//  }


}



template
void GPUGenPhi<float>(const float alpha,
                      const float sqrt_one_minus_alpha,
                      const float denom,
                      const float *waw_matrix_d,
                      const float *waf_matrix_d,
                      const float *faf_matrix_d,
                      const float *gamma_matrix_d,
                      const int n,
                      const int d,
                      const bool w_l_changed,
                      float *phi_of_alphas_d,
                      float *phi_of_zeros_d,
                      float *phi_of_zero_primes_d);


}  // Namespace NICE