/*
 * SVD.cu
 *
 *  Created on: Jun 16, 2016
 *      Author: cpurc002
 */
#include "gpu_svd_solver.h"
#include "hip/hip_runtime.h"
#include ""
#include "include/matrix.h"
#include "include/verctor.h" 
#include "include/gpu_util.h"
#include "Eigen/Dense"
#include<unistd.h>
#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
#include<hipsolver.h>
#include<hip/hip_runtime_api.h>

namespace NICE{ 
	template<class T>
	void GpuSvdSolver<T>::Compute(const Matrix<T>& A){
	
		// Initilize generally needed and misc variables 
		int work_size = 0;		
		int M = A.rows();
    		int N = A.cols();
		int *devInfo;   gpuErrchk(hipMalloc(&devInfo,          sizeof(int)));	
		float *work;    gpuErrchk(hipMalloc(&work, work_size * sizeof(float)));
	
		// Allocate all host and deviec memories 
		T *h_A = 	(T *)malloc(M * N * sizeof(T)); 
		T *h_U = 	(T *)malloc(M * M * sizeof(T));
    		T *h_V = 	(T *)malloc(N * N * sizeof(T));
    		T *h_S = 	(T *)malloc(N *     sizeof(T));
		T *d_A;         gpuErrchk(hipMalloc(&d_A,      M * N * sizeof(T)));
    		T *d_U;         gpuErrchk(hipMalloc(&d_U,      M * M * sizeof(T)));
    		T *d_V;         gpuErrchk(hipMalloc(&d_V,      N * N * sizeof(T)));
    		T *d_S;         gpuErrchk(hipMalloc(&d_S,      N *     sizeof(T)));

		// Map Eigan Matrix A to host matrix h_A and transfer to device matrix d_A
		Map<Matrix<T> >( h_A, M, N ) = A;
		cgpuErrchk(hipMemcpy(d_A, h_A, M * N * sizeof(T), hipMemcpyHostToDevice));
	
		// Initilize cuSolver 
		hipsolverStatus_t stat;	
		hipsolverHandle_t solver_handle;
		hipsolverDnCreate(&solver_handle);	
		stat = hipsolverDnSgesvd_bufferSize(solver_handle, M, N, &work_size);
		if(stat != HIPSOLVER_STATUS_SUCCESS ) std::cout << "Initialization of cuSolver failed. \N";
		
		// Execute and check status of SVD of A
	    	stat = hipsolverDnSgesvd(solver_handle, 'A', 'A', M, N, d_A, M, d_S, d_U, M, d_V, N, work, work_size, NULL, devInfo);
		
		switch(stat){
        		case HIPSOLVER_STATUS_SUCCESS:           std::cout << "SVD computation success\n";                       break;
        		case HIPSOLVER_STATUS_NOT_INITIALIZED:   std::cout << "Library cuSolver not initialized correctly\n";    break;
       		 	case HIPSOLVER_STATUS_INVALID_VALUE:     std::cout << "Invalid parameters passed\n";                     break;
        		case HIPSOLVER_STATUS_INTERNAL_ERROR:    std::cout << "Internal operation failed\n";                     break;
    		}
		
		// Copy device matrices to host matrices 
		gpuErrchk(hipMemcpy(h_S, d_S, 1 * N * sizeof(float), hipMemcpyDeviceToHost));
    		gpuErrchk(hipMemcpy(h_U, d_U, M * M * sizeof(float), hipMemcpyDeviceToHost));
	    	gpuErrchk(hipMemcpy(h_V, d_V, N * N * sizeof(float), hipMemcpyDeviceToHost));
		
		// Map host matrices to data member Eigan matrices 
                this->SingularValues = Map<Matrix<T> >(h_S, 1, N);
		this->MatrixU = Map<Matrix<T> >(h_U, M, M);
                this->MatrixV = Map<Matrix<T> >(h_V, N, N);

		
	}
}


