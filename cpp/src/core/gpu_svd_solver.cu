/*
 * GpuSvdSolver.cu
 *
 *  Created on: Jun 16, 2016
 *      Author: cpurc002
 */

#include "include/gpu_svd_solver.h"

namespace Nice {
		
template<typename T>
void GpuSvdSolver<T>::Compute(const Matrix<T> &A){

    	int M = A.rows();
    	int N = A.cols();
    	const T *h_A = &A(0); 

    	// --- Setting the device matrix and moving the host matrix to the device 
    	T *d_A;         gpuErrchk(hipMalloc(&d_A,      M * N * sizeof(T)));
    	gpuErrchk(hipMemcpy(d_A, h_A, M * N * sizeof(T), hipMemcpyHostToDevice));

   	// --- host side SVD results space 
    	s_.resize(M,1); 
    	u_.resize(M,M); 
    	v_.resize(N,N); 

    	// --- device side SVD workspace and matrices 
    	int work_size = 0;
    	int *devInfo;       gpuErrchk(hipMalloc(&devInfo,          sizeof(int)));
    	T *d_U;         gpuErrchk(hipMalloc(&d_U,      M * M * sizeof(T)));
    	T *d_V;         gpuErrchk(hipMalloc(&d_V,      N * N * sizeof(T)));
    	T *d_S;         gpuErrchk(hipMalloc(&d_S,      N *     sizeof(T)));

    	hipsolverStatus_t stat;
    	// --- CUDA solver initialization
    	hipsolverHandle_t solver_handle;
    	hipsolverDnCreate(&solver_handle);
    	stat = hipsolverDnSgesvd_bufferSize(solver_handle, M, N, &work_size);
    	if(stat != HIPSOLVER_STATUS_SUCCESS ) std::cout << "Initialization of cuSolver failed. \n";
    	T *work;    gpuErrchk(hipMalloc(&work, work_size * sizeof(T)));

    	// --- CUDA SVD execution
    	stat = hipsolverDnSgesvd(solver_handle, 'A', 'A', M, N, d_A, M, d_S, d_U, M, d_V, N, work, work_size, NULL, devInfo);
    	hipDeviceSynchronize();

    	int devInfo_h = 0;
    	gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    	std::cout << "devInfo = " << devInfo_h << "\n";

    	switch(stat){
        	case HIPSOLVER_STATUS_SUCCESS:           std::cout << "SVD computation success\n";                       break;
        	case HIPSOLVER_STATUS_NOT_INITIALIZED:   std::cout << "Library cuSolver not initialized correctly\n";    break;
        	case HIPSOLVER_STATUS_INVALID_VALUE:     std::cout << "Invalid parameters passed\n";                     break;
        	case HIPSOLVER_STATUS_INTERNAL_ERROR:    std::cout << "Internal operation failed\n";                     break;
        	default: break;
        }
    
    	if (devInfo_h == 0 && stat == HIPSOLVER_STATUS_SUCCESS) std::cout    << "SVD successful\n\n";
    	std::cout<<std::endl;

    	// --- Moving the results from device to host
    	gpuErrchk(hipMemcpy(&s_(0,0), d_S, N * sizeof(T), hipMemcpyDeviceToHost));
    	gpuErrchk(hipMemcpy(&u_(0,0), d_U, M * M * sizeof(T), hipMemcpyDeviceToHost));
    	gpuErrchk(hipMemcpy(&v_(0,0), d_V, N * N * sizeof(T), hipMemcpyDeviceToHost));

	hipFree(d_S); hipFree(d_U); hipFree(d_V); 
   	hipsolverDnDestroy(solver_handle);
}

	template class GpuSvdSolver<float>;
//	template class GpuSvdSolver<double>;

}


