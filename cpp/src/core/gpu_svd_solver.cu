/*
 * GpuSvdSolver.cu
 *
 *  Created on: Jun 16, 2016
 *      Author: cpurc002
 */

#include "include/gpu_svd_solver.h"

/*
#include<hip/hip_runtime.h>
#include ""
#include "include/matrix.h"
#include "include/vector.h"
#include "include/gpu_util.h"
#include "Eigen/Dense"
#include<unistd.h>
#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
#include<hipsolver.h>
#include<hip/hip_runtime_api.h>


*/
namespace Nice {
/*
template<typename T>
class GpuSvdSolver {
 public:
   GpuSvdSolver();
   void Compute(const Matrix<T> &A);
   Matrix<T> MatrixU() const;
   Matrix<T> MatrixV() const;
   Vector<T> SingularValues() const;
};
*/
	template<typename T>
	void GpuSvdSolver<T>::Compute(const Matrix<T>& A){
		//------------------------------------------------------------------------
		// TO BE DONE
		// cuSolver cant take ints, only float doublt hipComplex cuComplexDouble
		//------------------------------------------------------------------------		
//		int i
		// Initilize generally needed and misc variables 
		int work_size = 0;		
		int M = A.rows();
    		int N = A.cols();
		int *devInfo;   gpuErrchk(hipMalloc(&devInfo,          sizeof(int)));	
		float *work;    gpuErrchk(hipMalloc(&work, work_size * sizeof(float)));
			
		// Allocate all host and deviec memories 
		T *h_A = 	(T *)malloc(M * N * sizeof(T)); 
		T *h_U = 	(T *)malloc(M * M * sizeof(T));
    		T *h_V = 	(T *)malloc(N * N * sizeof(T));
    		T *h_S = 	(T *)malloc(N *     sizeof(T));
		T *d_A;         gpuErrchk(hipMalloc(&d_A,      M * N * sizeof(T)));
    		T *d_U;         gpuErrchk(hipMalloc(&d_U,      M * M * sizeof(T)));
    		T *d_V;         gpuErrchk(hipMalloc(&d_V,      N * N * sizeof(T)));
    		T *d_S;         gpuErrchk(hipMalloc(&d_S,      N *     sizeof(T)));

		// Map Eigan Matrix A to host matrix h_A and transfer to device matrix d_A

		gpuErrchk(hipMemcpy(d_A, h_A, M * N * sizeof(T), hipMemcpyHostToDevice));
	
		// Initilize cuSolver 
		hipsolverStatus_t stat;	
		hipsolverHandle_t solver_handle;
		hipsolverDnCreate(&solver_handle);	
		stat = hipsolverDnSgesvd_bufferSize(solver_handle, M, N, &work_size);
		if(stat != HIPSOLVER_STATUS_SUCCESS ) std::cout << "Initialization of cuSolver failed. \n";
		
		// Execute and check status of SVD of A
	    	stat = hipsolverDnSgesvd(solver_handle, 'A', 'A', M, N, d_A, M, d_S, d_U, M, d_V, N, work, work_size, NULL, devInfo);
		
		switch(stat){
        		case HIPSOLVER_STATUS_SUCCESS:           std::cout << "SVD computation success\n";                       break;
        		case HIPSOLVER_STATUS_NOT_INITIALIZED:   std::cout << "Library cuSolver not initialized correctly\n";    break;
       		 	case HIPSOLVER_STATUS_INVALID_VALUE:     std::cout << "Invalid parameters passed\n";                     break;
        		case HIPSOLVER_STATUS_INTERNAL_ERROR:    std::cout << "Internal operation failed\n";                     break;
			default: break; 
    		}
		
		// Copy device matrices to host matrices 
		gpuErrchk(hipMemcpy(h_S, d_S, 1 * N * sizeof(T), hipMemcpyDeviceToHost));
    		gpuErrchk(hipMemcpy(h_U, d_U, M * M * sizeof(T), hipMemcpyDeviceToHost));
	    	gpuErrchk(hipMemcpy(h_V, d_V, N * N * sizeof(T), hipMemcpyDeviceToHost));
		
	
	}
	
//	template class GpuSvdSolver<int>;
	template class GpuSvdSolver<float>;
//	template class GpuSvdSolver<double>;

}


